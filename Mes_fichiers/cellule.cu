#include "hip/hip_runtime.h"
#include "gpu_bitmap.h"
#include "gpu_bitmap.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define WIDTH 800
#define HEIGHT 600
#define DIM 16

#define K 3
#define G 28
#define M 200


__global__ void color(float *t, uchar4 *buf) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		float t3 = 3 * t[offset];
		float rouge, vert, blue;
		if (t3 == 1) {
			rouge = t3 * 255;vert= 2;blue = 0;
		} else {
			rouge = 255;vert = (t3 - 1) * 255;blue = 0;
		}
		buf[offset].x = rouge;
		buf[offset].y = vert;
		buf[offset].z = blue;
		buf[offset].w = 255;
	}
}

__global__ void diffuse(float *t_current, float *t_next) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;
	if (x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		int haut = y == HEIGHT - 1 ? offset : offset + WIDTH;
		int haut_gauche;
		
		if (y == HEIGHT - 1 || x == 0 )haut_gauche = offset;
		else haut_gauche = offset + WIDTH;

	  	int gauche = x == 0 ? offset : offset - 1;
    		int bas_gauche = (x == 0 || y == 0) ? offset : offset - WIDTH - 1;
		int bas = y == 0 ? offset : offset - WIDTH;
		int bas_droite = (y == 0 || x == WIDTH -1) ? offset : offset - WIDTH + 1;
		int droite = x == WIDTH - 1 ? offset : offset + 1;
		int haut_droite  = (y == 0 || x == WIDTH -1 ) ? offset : offset - WIDTH + 1;


		int vie = (t_current[haut] == 0 ? 0:1) +(t_current[bas] == 0 ? 0:1)+
		(t_current[gauche] == 0 ? 0:1)+(t_current[droite] == 0 ? 0:1) +
		(t_current[haut_gauche] == 0 ? 0:1) +
		(t_current[haut_droite] == 0 ? 0:1) + (t_current[bas_gauche] == 0 ? 0:1)+(t_current[bas_droite] == 0 ? 0:1) ;
				if(t_current[offset] == 0){
					    if(vie == 3)t_next[offset] = 1;
						else t_next[offset] = 0;			
				}
		    	if(t_current[offset] == 1){
						if(vie == 2 || vie == 3)t_next[offset] = 1;
						else t_next[offset] = 0;		
				}
		t_next[offset] = (1 - 4 * K) * t_current[offset] + K * (t_current[haut] + t_current[bas] + t_current[gauche] + t_current[droite]);
	}
}

struct Donnee {
	float *t1,*t2;
	dim3 blocks,threads;
};
void clean_callback(Donnee *donnee) {
	HANDLE_CUDA_ERR(hipFree(donnee->t1));
	HANDLE_CUDA_ERR(hipFree(donnee->t2));
}

void render_callback(uchar4 *buf, Donnee *donnee, int ticks) {
	diffuse<<<donnee->blocks, donnee->threads>>>(donnee->t1, donnee->t2);
	diffuse<<<donnee->blocks, donnee->threads>>>(donnee->t2, donnee->t1);
	color<<<donnee->blocks, donnee->threads>>>(donnee->t1, buf);

}

int main() {
	Donnee donnee;
	GPUBitmap bitmap(WIDTH, HEIGHT, &donnee, "Automate cellulaire");

	size_t taille = WIDTH * HEIGHT * sizeof(float);
	float *t_initial = (float *)calloc(WIDTH * HEIGHT, sizeof(float));
	srand(time(NULL));

	for (int x = 0; x < WIDTH; x++) {
		for (int y = 0; y < HEIGHT; y++) {
				int r = rand();
				if(r%2 == 0)t_initial[y * WIDTH + x] = 1;
				else t_initial[y * WIDTH + x] = 0;
		}

	}
	donnee.blocks = dim3((WIDTH + DIM - 1) / DIM, (HEIGHT + DIM - 1) / DIM);
	donnee.threads = dim3(DIM, DIM);
	HANDLE_CUDA_ERR(hipMalloc(&donnee.t1, taille));
	HANDLE_CUDA_ERR(hipMalloc(&donnee.t2, taille));
	HANDLE_CUDA_ERR(hipMemcpy(donnee.t1, t_initial, taille, hipMemcpyHostToDevice));
	bitmap.animate((void (*)(uchar4*, void*, int))render_callback, (void (*)(void*))clean_callback);
	return 0;
}
